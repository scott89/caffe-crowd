#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void KLDivergenceLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->cpu_data();
  const Dtype* target = bottom[1]->cpu_data();
  Dtype loss = 0;
  for (int i = 0; i < count; ++i) {
    loss += target[i] * log(target[i] / (input_data[i] + (target[i] == Dtype(0)))) +
        (1-target[i]) * log((1-target[i]) / (1-input_data[i] + (target[i] == Dtype(1))));
  }
  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
void KLDivergenceLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
    << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const Dtype* input_data = bottom[0]->cpu_data();
    const Dtype* target = bottom[1]->cpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
    for (int i = 0; i < count; i++) {
      bottom_diff[i] = - target[i] / (input_data[i] + (target[i] == Dtype(0))) +
        (1-target[i]) / (1-input_data[i] + (target[i] == Dtype(1)));
    }
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_scal(count, loss_weight / num, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KLDivergenceLossLayer);
  
}  // namespace caffe